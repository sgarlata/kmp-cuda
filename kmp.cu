/*
CUDA-BASED IMPLEMENTATION OF THE KMP ALGORITHM
VERSION: USING ALSO THE GPU (NOT POLISHED)
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>

#define MAX 50

__managed__ int match = 0;

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

void computeNext(int *next, char *pattern, int m) {
  int j = 1, t = 0;
  next[0] = -1;

  while (j < m) {
    while (t > 0 && pattern[j] != pattern[t])
      t = next[t];
    ++t;
    ++j;

    if (pattern[j] == pattern[t])
      next[j] = next[t];
    else
      next[j] = t;
  }
}

__global__ void patternMatch(char *pattern, char *text, int *next, int m, int n) {
  int j; // current position in pattern
  int k; // current position in text
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int sublength = ceilf( (float) n / (gridDim.x * blockDim.x));
  int start = idx * sublength;
  int stop = start + sublength;
  int proceed = 1;

  printf("Thread: %d. Start: %d. Stop: %d.\n", idx, start, stop);

  for (j = 0, k = start; j < m && k < n && proceed; ++j, ++k) {
    //printf("Thread: %d. Inside for with j = %d and k = %d\n", idx, j, k);
    while (j >= 0 && text[k] != pattern[j]) {
      //printf("Thread: %d. Inside while with j = %d and k = %d\n", idx, j, k);
      j = next[j];
      //printf("Thread: %d. j is now %d\n", idx, j);
      if (k - j >= stop) {
      //printf("Thread: %d. Inside if with k = %d.\n", idx, k);
        proceed = 0;
        }
    }
  }

    if (j == m) {
      match = 1;
      printf("Match found by thread n. %d in positions %d through %d.\n", idx, k - m, k - 1);
    }
}

int kmp(char *text, char *pattern, int N, int M) {
  int *next;
  size_t next_size = M * sizeof(int);

  checkCuda(hipMallocManaged(&next, next_size));
  computeNext(next, pattern, M);

  size_t threads_per_block = 4;
  size_t number_of_blocks = 2;

  patternMatch<<<number_of_blocks, threads_per_block>>> (pattern, text, next, M, N);
  checkCuda(hipGetLastError());
  
  checkCuda(hipDeviceSynchronize());

  return match;
};

int main() {
  char buffer[MAX+1], *text, *pattern;
  int N, M;

  printf("Enter text (at most %d characters): ", MAX);
  fgets(buffer, MAX+1, stdin);
  buffer[strcspn(buffer, "\n")] = 0; // to remove \n
  N = strlen(buffer);
  size_t text_size = (N + 1) * sizeof(char);
  checkCuda(hipMallocManaged(&text, text_size));
  strncpy(text, buffer, N);

  printf("Enter pattern: (at most %d characters): ", MAX);
  fgets(buffer, MAX+1, stdin);
  buffer[strcspn(buffer, "\n")] = 0; // to remove \n
  M = strlen(buffer);
  size_t pattern_size = (M + 1) * sizeof(char);
  checkCuda(hipMallocManaged(&pattern, pattern_size));
  strncpy(pattern, buffer, M);  

  if (!kmp(text, pattern, N, M))
    printf("No match found.\n");
    
  checkCuda(hipFree(text));
  checkCuda(hipFree(pattern));

  return 0;
}