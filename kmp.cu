/*
CUDA-BASED IMPLEMENTATION OF THE KMP ALGORITHM

THIS VERSION CAN READ THE INPUT TEXT FROM A FILE.
THE CURRENT APPROACH IS TO WORK LINE BY LINE.
AN ALTERNATIVE ONE COULD BE TO CONCATENATE ALL THE LINES
AND THEN WORK AS BEFORE ON THE RESULTING SINGLE BIG STRING.
TODO: FIGURE OUT WHICH IS THE BEST APPROACH
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>

#define MAX 50

__managed__ int match = 0; // to know whether at least a match in the whole file was found

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

void computeNext(int *next, char *pattern, int M) {
  int j = 1, t = 0;
  next[0] = -1;

  while (j < M) {
    while (t > 0 && pattern[j] != pattern[t])
      t = next[t];
    ++t;
    ++j;

    if (pattern[j] == pattern[t])
      next[j] = next[t];
    else
      next[j] = t;
  }
}

__global__ void patternMatch(char *pattern, char *text, int *next, int *matchedText, int M, int N) {
  int j; // current position in pattern
  int k; // current position in text
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int sublength = ceilf( (float) N / (gridDim.x * blockDim.x)); // text characters divided by the number of threads
  int start = idx * sublength; // initial delimiter for each thread (included)
  int stop = start + sublength; // final delimiter for each thread (excluded)
  int proceed = 1;

  for (j = 0, k = start; j < M && k < N && proceed; ++j, ++k) {
    while (j >= 0 && text[k] != pattern[j]) {
      j = next[j];
      if (k - j >= stop)
        proceed = 0; // a match should only be found by a single thread, namely the one whose text's portion contains the initial character of the matched sequence
    }
  }

  if (j == M) { // a match was found
    match = 1;
    matchedText[k - M] = k - 1;
  }
}

int checkMatch(char *pattern, char *text, int M, int N, int start, int end) {
  int check = 1;

  if (end - start + 1 != M) // wrong length
    check = 0;

  for (int i = start; i < M && i <= end; ++i)
    if (pattern[i] != text[i]) // wrong character
      check = 0;

  return check;
}

void kmp(char *text, char *pattern, int N, int M, int line) {
  int *next; // auxiliary array to know how far to slide the pattern when a mismatch is detected
  int *matchedText; // array to store the matched text's positions
  int i; // used to loop

  checkCuda(hipMallocManaged(&next, M * sizeof(int)));
  computeNext(next, pattern, M);

  checkCuda(hipMallocManaged(&matchedText, N * sizeof(int)));
  for (i = 0; i < N; ++i)
    matchedText[i] = -1;
  
  size_t threads_per_block = 4;
  size_t number_of_blocks = 2;

  patternMatch<<<number_of_blocks, threads_per_block>>> (pattern, text, next, matchedText, M, N);
  checkCuda(hipGetLastError());
  
  checkCuda(hipDeviceSynchronize());

  checkCuda(hipFree(next));

  for (i = 0; i < N; ++i)
    if (matchedText[i] != -1)
      if (checkMatch(pattern, text, M, N, i, matchedText[i])) // we check the match for correctness
        printf("Match found on line %d from position %d through %d\n", line, i + 1, matchedText[i] + 1);
}

int main(int argc, char *argv[]) {
  FILE *fp;
  char buffer[MAX+1], *text, *pattern;
  int N, M, line;

  if (argc < 3) {
    printf("You must provide the name of the text file as the first argument and the pattern as the second one\n");
    return EXIT_FAILURE;
  }

  fp = fopen(argv[1], "r");
  if (fp == NULL) {
    printf("Error with file\n");
    return EXIT_FAILURE;
  }

  M = strlen(argv[2]);
  checkCuda(hipMallocManaged(&pattern, (M + 1) * sizeof(char)));
  strncpy(pattern, argv[2], M);

  line = 0;
  while (fgets(buffer, MAX+1, fp) != NULL) { // we apply kmp to each line of the file
    buffer[strcspn(buffer, "\n")] = 0; // to remove \n
    N = strlen(buffer);
    checkCuda(hipMallocManaged(&text, (N + 1) * sizeof(char)));
    strncpy(text, buffer, N);

    kmp(text, pattern, N, M, ++line);
    
    checkCuda(hipFree(text));
  }

  checkCuda(hipFree(pattern));

  if (!match)
    printf("No match was found.\n");

  return 0;
}